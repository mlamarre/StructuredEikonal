#include "hip/hip_runtime.h"
//
// CUDA implementation of FIM (Fast Iterative Method) for Eikonal equations
//
// Copyright (c) Won-Ki Jeong (wkjeong@unist.ac.kr)
//
// 2016. 2. 4
//

#include <cstdio>
#include <string>
#include <cmath>
#include <float.h>
#include <assert.h>
#include <vector>
#include <iostream>
#include "cuda_fim_kernel.h"
#include "cuda_fim.h"

void CUT_SAFE_CALL(hipError_t error) {
	if(error != hipSuccess)
	{
		printf("CUDA error! %d \n",error);
		exit(EXIT_FAILURE);
	}
}

void CUDA_SAFE_CALL(hipError_t error) {
	CUT_SAFE_CALL(error);
}

void runEikonalSolverSimple(CUDAMEMSTRUCT &cmem, bool verbose)
{
  int deviceID;
  hipGetDevice(&deviceID);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceID);
  if (verbose) {
    printf("Current device id : %d, name : %s\n", deviceID, deviceProp.name);
  }

  int xdim, ydim, zdim;
  xdim = cmem.xdim;
  ydim = cmem.ydim;
  zdim = cmem.zdim;

#ifdef __DEVICE_EMULATION__
  assert(xdim%BLOCK_LENGTH == 0);
  assert(ydim%BLOCK_LENGTH == 0);
  assert(zdim%BLOCK_LENGTH == 0);
#endif

  // create volumes
  uint volSize = cmem.volsize;
  uint blockNum = cmem.blknum;

  //int nBlkZ = zdim/BLOCK_LENGTH;

  if (verbose) {
    printf("# of total voxels : %d\n", volSize);
    printf("# of total blocks : %d\n", blockNum);
  }

  // h_ : host memory, d_ : device memory

  // copy speed table to constant variable
  //hipMemcpyToSymbol((const char*)_speed, cmem.h_speedtable, sizeof(float)*256);

  int nIter = cmem.nIter;
  uint nActiveBlock = cmem.nActiveBlock; // active list

  double *d_spd = cmem.d_spd;
  DOUBLE *d_sol = cmem.d_sol;
  DOUBLE *t_sol = cmem.t_sol;

  uint *d_list = cmem.d_list;
  bool *d_listVol = cmem.d_listVol;

  bool *d_con = cmem.d_con;
  bool *d_mask = cmem.d_mask;

  // copy so that original value should not be modified
  uint *h_list = (uint*) malloc(blockNum*sizeof(uint));
  bool *h_listed = (bool*) malloc(blockNum*sizeof(bool));
  bool *h_listVol = (bool*) malloc(blockNum*sizeof(bool));

  // initialization
  memcpy(h_list, cmem.h_list, blockNum*sizeof(uint));
  memcpy(h_listed, cmem.h_listed, blockNum*sizeof(bool));
  memcpy(h_listVol, cmem.h_listVol, blockNum*sizeof(bool));

  CUT_SAFE_CALL( hipMemcpy(cmem.d_list, cmem.h_list, nActiveBlock*sizeof(uint), hipMemcpyHostToDevice) );
  CUT_SAFE_CALL( hipMemcpy(cmem.d_listVol, cmem.h_listVol, blockNum*sizeof(bool), hipMemcpyHostToDevice) );
  CUT_SAFE_CALL( hipMemcpy(cmem.d_sol, cmem.h_sol, volSize*sizeof(DOUBLE), hipMemcpyHostToDevice) );
  CUT_SAFE_CALL( hipMemcpy(cmem.t_sol, cmem.h_sol, volSize*sizeof(DOUBLE), hipMemcpyHostToDevice) );
  CUT_SAFE_CALL( hipMemset(cmem.d_con, 1, volSize*sizeof(bool)) );

  // set dimension of block and entire grid size
  dim3 dimBlock(BLOCK_LENGTH,BLOCK_LENGTH,BLOCK_LENGTH);
  dim3 dimEntireGrid(blockNum);
  dim3 dimGrid(nActiveBlock);

  int nTotalIter = 0;
  //uint sharedmemsize = sizeof(float)*BLOCK_LENGTH*BLOCK_LENGTH*(3*BLOCK_LENGTH + 2);

  std::vector<int> sourceList;
  sourceList.push_back((zdim/2)*ydim*xdim + (ydim/2)*xdim + (xdim/2));

  // initialize & start timer
  StopWatchInterface *timer_total, *timer_solver, *timer_reduction, *timer_list, *timer_list2, *timer_coarse;
  timer_total = timer_solver = timer_reduction = timer_list = timer_list2 = timer_coarse = NULL;

  sdkCreateTimer(&timer_total);
  sdkCreateTimer(&timer_solver);
  sdkCreateTimer(&timer_reduction);
  sdkCreateTimer(&timer_list);
  sdkCreateTimer(&timer_list2);
  sdkCreateTimer(&timer_coarse);

  sdkStartTimer(&timer_total);

  uint nTotalBlockProcessed = 0;

  // start solver
  while(nActiveBlock > 0)
  {
    //CUT_SAFE_CALL( hipMemcpy(cmem.d_sol, cmem.h_sol, 64*nActiveBlock*sizeof(DOUBLE), hipMemcpyHostToDevice) );
    assert(nActiveBlock < 4294967295);

    nTotalBlockProcessed += nActiveBlock;

    nTotalIter++;

    //
    // solve current blocks in the active lists
    //

    //      printf("# of active tiles : %u\n", nActiveBlock);
    if (verbose) {
      printf("# of active tiles : %u\n", nActiveBlock);
    }
    //////////////////////////////////////////////////////////////////
    // 1. run solver on current active tiles

    sdkStartTimer(&timer_solver);

    dimGrid.y = (unsigned int)floor(((double)nActiveBlock-1)/65535)+1;
    dimGrid.x = (unsigned int)ceil ((double)nActiveBlock/(double)dimGrid.y);

    if (verbose) {
      printf("Grid size : %d x %d\n", dimGrid.x, dimGrid.y);
    }

    CUT_SAFE_CALL( hipMemcpy(d_list, h_list, nActiveBlock*sizeof(uint), hipMemcpyHostToDevice) );

    run_solver<<< dimGrid, dimBlock >>>(d_spd, d_mask, d_sol, t_sol, d_con, d_list, xdim, ydim, zdim, nIter, nActiveBlock);

    CUT_SAFE_CALL(hipGetLastError());

    hipDeviceSynchronize();
    sdkStopTimer(&timer_solver);


    //////////////////////////////////////////////////////////////////
    // 2. reduction (only active tiles)

    sdkStartTimer(&timer_reduction);

    run_reduction<<< dimGrid, dim3(BLOCK_LENGTH,BLOCK_LENGTH,BLOCK_LENGTH/2) >>>(d_con, d_listVol, d_list, nActiveBlock);

    CUT_SAFE_CALL(hipGetLastError());
    //CUT_CHECK_ERROR("Kernel execution failed");
    hipDeviceSynchronize();

    sdkStopTimer(&timer_reduction);


    //////////////////////////////////////////////////////////////////
    // 3. check neighbor tiles of converged tile
    // Add any active block of neighbor of converged block is inserted
    // to the list

    // read-back active list volume
#ifdef TIMER
    sdkStartTimer(&timer_list);
#endif
    CUT_SAFE_CALL( hipMemcpy(h_listVol, d_listVol, blockNum*sizeof(bool), hipMemcpyDeviceToHost) );

    uint nOldActiveBlock = nActiveBlock;
    uint nBlkX = xdim/BLOCK_LENGTH;
    uint nBlkY = ydim/BLOCK_LENGTH;

    for(uint i=0; i<nOldActiveBlock; i++)
    {
      // check 6-neighbor of current active tile
      uint currBlkIdx = h_list[i];

      if(!h_listVol[currBlkIdx]) // not active : converged
      {
        uint nb[6];
        nb[0] = (currBlkIdx < nBlkX*nBlkY) ? currBlkIdx : (currBlkIdx - nBlkX*nBlkY);  //tp
        nb[1] = ((currBlkIdx + nBlkX*nBlkY) >= blockNum) ? currBlkIdx : (currBlkIdx + nBlkX*nBlkY); //bt
        nb[2] = (currBlkIdx < nBlkX) ? currBlkIdx : (currBlkIdx - nBlkX); //up
        nb[3] = ((currBlkIdx + nBlkX) >= blockNum) ? currBlkIdx : (currBlkIdx + nBlkX); //dn
        nb[4] = (currBlkIdx%nBlkX == 0) ? currBlkIdx : currBlkIdx-1; //lf
        nb[5] = ((currBlkIdx+1)%nBlkX == 0) ? currBlkIdx : currBlkIdx+1; //rt

        for(int nbIdx = 0; nbIdx < 6; nbIdx++)
        {
          uint currIdx = nb[nbIdx];

          //  assert(currIdx < volSize);

          if(!h_listed[currIdx])
          {
            h_listed[currIdx] = true;
            h_list[nActiveBlock++] = currIdx;
          }
        }
      }
    }
    hipDeviceSynchronize();
#ifdef TIMER
    sdkStopTimer(&timer_list);
#endif


    //////////////////////////////////////////////////////////////////
    // 4. run solver only once for neighbor blocks of converged block
    // current active list contains active blocks and neighbor blocks of
    // any converged blocks.
    //

#ifdef TIMER
    sdkStartTimer(&timer_solver);
#endif

    // update grid dimension because nActiveBlock is changed
    dimGrid.y = (unsigned int)floor(((double)nActiveBlock-1)/65535)+1;
    dimGrid.x = (unsigned int)ceil((double)nActiveBlock/(double)dimGrid.y);

    if (verbose) {
      printf("Grid size : %d x %d\n", dimGrid.x, dimGrid.y);
    }

    CUT_SAFE_CALL(hipMemcpy(d_list, h_list, nActiveBlock*sizeof(uint), hipMemcpyHostToDevice) );
    run_check_neighbor<<< dimGrid, dimBlock >>>(d_spd, d_mask, t_sol, d_sol, d_con, d_list, xdim, ydim, zdim, nOldActiveBlock, nActiveBlock);
    CUT_SAFE_CALL(hipGetLastError());
    hipDeviceSynchronize();

#ifdef TIMER
    sdkStopTimer(&timer_solver);
#endif



    //////////////////////////////////////////////////////////////////
    // 5. reduction

    sdkStartTimer(&timer_reduction);

    run_reduction<<< dimGrid, dim3(BLOCK_LENGTH,BLOCK_LENGTH,BLOCK_LENGTH/2) >>>(d_con, d_listVol, d_list, nActiveBlock);
    CUT_SAFE_CALL(hipGetLastError());
    hipDeviceSynchronize();

    sdkStopTimer(&timer_reduction);


    //////////////////////////////////////////////////////////////////
    // 6. update active list
    // read back active volume from the device and add
    // active block to active list on the host memory


#ifdef TIMER
    sdkStartTimer(&timer_list2);
#endif
    nActiveBlock = 0;
    CUT_SAFE_CALL( hipMemcpy(h_listVol, d_listVol, blockNum*sizeof(bool), hipMemcpyDeviceToHost) );

    for(uint i=0; i<blockNum; i++)
    {
      if(h_listVol[i]) // true : active block (not converged)
      {
        h_listed[i] = true;
        h_list[nActiveBlock++] = i;
        //printf("Block %d added\n", i);
      }
      else h_listed[i] = false;
    }
    hipDeviceSynchronize();
#ifdef TIMER
    sdkStopTimer(&timer_list2);
#endif


    if (verbose) {
      printf("Iteration : %d\n", nTotalIter);
    }
  }
  sdkStopTimer(&timer_total);

  if (verbose) {
    printf("Eikonal solver converged after %d iterations\n", nTotalIter);
    printf("Total Running Time: %f (sec)\n", sdkGetTimerValue(&timer_total) / 1000);
    printf("Time for solver : %f (sec)\n", sdkGetTimerValue(&timer_solver) / 1000);
    printf("Time for reduction : %f (sec)\n", sdkGetTimerValue(&timer_reduction) / 1000);
#ifdef TIMER
    printf("Time for list update-1 (CPU) : %f (sec)\n", sdkGetTimerValue(&timer_list) / 1000);
    printf("Time for list update-2 (CPU) : %f (sec)\n", sdkGetTimerValue(&timer_list2) / 1000);

#endif
    printf("Total # of blocks processed : %d\n", nTotalBlockProcessed);
  }

  sdkDeleteTimer(&timer_total);
  sdkDeleteTimer(&timer_solver);
  sdkDeleteTimer(&timer_reduction);
  sdkDeleteTimer(&timer_list);

  // delete dynamically allocated host memory
  free(h_list);
  free(h_listed);
  free(h_listVol);
}
